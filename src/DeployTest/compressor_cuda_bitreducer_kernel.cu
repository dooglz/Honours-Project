#include "hip/hip_runtime.h"
#include <stdint.h>

__global__ void bit_reduce(const uint32_t *input_array, const uint16_t dataSize, uint32_t *intBuf)
{
 uint32_t i = threadIdx.x + blockDim.x * blockIdx.x;
 uint32_t a = input_array[i];

 if (a <= 0xff) {
	 intBuf[i] = 1;
	 uint8_t b = static_cast<uint8_t>(a);
	 memcpy((uint8_t*)(&intBuf[i]) + 1, &b, sizeof(uint8_t));
 }
 else if (a <= 0xffff) {
	 intBuf[i] = sizeof(uint16_t);
	 uint16_t s = static_cast<uint16_t>(a);
	 memcpy((uint8_t*)(&intBuf[i]) + 1, &s, sizeof(uint16_t));
 }
 else {
	 intBuf[i] = sizeof(uint32_t);
	 memcpy((uint8_t*)(&intBuf[i]) + 1, &a, sizeof(uint32_t));
 }

}


void Run_BitReduce(const dim3 a, const dim3 b, const uint32_t *input_array, const uint16_t dataSize, uint32_t *intBuf) {
	bit_reduce << <a, b >> > (input_array, dataSize, intBuf);
	//hipDeviceSynchronize();
}
